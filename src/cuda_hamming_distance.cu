#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

namespace dce {
        namespace metrics {
            namespace cuda {
                namespace hamming {

                template<typename TValue>
                __global__ void distance(size_t size, TValue *vec_1, TValue *vec_2, TValue *distance) {
                    int index = threadIdx.x + blockIdx.x * blockDim.x;

                    TValue count = 0.0f;

                    if (vec_1[index] != vec_2[index]) {
                        count++;
                    }

                    *distance = count / size;
                }

                template<typename TValue, typename TVectorIter>
                TValue cuda_distance(size_t size, TVectorIter iter1, TVectorIter iter2) {
                    TValue* host_vec_1, host_vec_2, value;
                    host_vec_1 = (TValue *)malloc(size);
                    host_vec_2 = (TValue *)malloc(size);
                    value = (TValue *)malloc(sizeof(TValue));


                    TValue *device_vec_1, device_vec_2, distance;
                    hipMalloc((void **) &device_vec_1, size);
                    hipMalloc((void **) &device_vec_1, size);
                    hipMalloc((void **) &distance, sizeof(TValue));

                    for (int i = 0; i < size; ++i) {
                        host_vec_1[i] = iter1[i];
                        host_vec_2[i] = iter2[i];
                    }

                    // Copy inputs to device
                    hipMemcpy(device_vec_1, host_vec_1, size, hipMemcpyHostToDevice);
                    hipMemcpy(device_vec_2, host_vec_2, size, hipMemcpyHostToDevice);

                    int THREADS_PER_BLOCK = 512;

                    distance < TValue ><<<size /
                                          THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(size, device_vec_1, device_vec_2, distance);

                    hipMemcpy(value, distance, sizeof(TValue), hipMemcpyDeviceToHost);


                    free(host_vec_1);
                    free(host_vec_2);
                    free(value);

                    hipFree(device_vec_1);
                    hipFree(device_vec_2);
                    hipFree(distance);

                    return *value;
                }
            }
        }
    }
}